#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

#define SIZE (1024 * 1024 * 10)  // 10 миллионов элементов
#define MAX_BLOCK_SIZE 1024
#define ITERATIONS 10

// Ядро для сложения векторов
__global__ void vector_add_kernel(float* A, float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) C[idx] = A[idx] + B[idx];
}

// Ядро для скалярного произведения с редукцией
__global__ void dot_product_kernel(float* A, float* B, float* result, int N) {
    __shared__ float cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    
    float temp = 0;
    while (tid < N) {
        temp += A[tid] * B[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    cache[cacheIndex] = temp;
    __syncthreads();
    
    // Редукция
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    
    if (cacheIndex == 0)
        atomicAdd(result, cache[0]);
}

// Тестирование скорости копирования
void benchmark_copy(size_t size, bool pinned) {
    float *h_data, *d_data;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    if (pinned) hipHostAlloc(&h_data, size, hipHostMallocDefault);
    else h_data = new float[size/4];
    
    hipMalloc(&d_data, size);
    
    // Host -> Device
    hipEventRecord(start);
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "H2D " << (pinned ? "pinned" : "normal") 
              << " time: " << ms << " ms" << std::endl;
    
    // Device -> Host
    hipEventRecord(start);
    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "D2H " << (pinned ? "pinned" : "normal") 
              << " time: " << ms << " ms" << std::endl;

    if (pinned) hipHostFree(h_data);
    else delete[] h_data;
    hipFree(d_data);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// Тестирование сложения векторов с разными блоками
void benchmark_vector_add() {
    const int N = 1024 * 1024 * 10;
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    
    hipHostAlloc(&h_A, N*sizeof(float), hipHostMallocDefault);
    hipHostAlloc(&h_B, N*sizeof(float), hipHostMallocDefault);
    hipHostAlloc(&h_C, N*sizeof(float), hipHostMallocDefault);
    
    hipMalloc(&d_A, N*sizeof(float));
    hipMalloc(&d_B, N*sizeof(float));
    hipMalloc(&d_C, N*sizeof(float));
    
    for (int block_size = 32; block_size <= MAX_BLOCK_SIZE; block_size *= 2) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        float total_time = 0;
        for (int i = 0; i < ITERATIONS; ++i) {
            hipMemcpy(d_A, h_A, N*sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_B, h_B, N*sizeof(float), hipMemcpyHostToDevice);
            
            hipEventRecord(start);
            int grid_size = (N + block_size - 1) / block_size;
            vector_add_kernel<<<grid_size, block_size>>>(d_A, d_B, d_C, N);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            total_time += ms;
        }
        std::cout << "Block size " << block_size 
                  << " average time: " << total_time/ITERATIONS << " ms" << std::endl;
        
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Тестирование скалярного произведения с разными блоками
void benchmark_dot_product() {
    const int N = 1024 * 1024 * 10;
    float *h_A, *h_B;
    float *d_A, *d_B, *d_result;
    
    hipHostAlloc(&h_A, N*sizeof(float), hipHostMallocDefault);
    hipHostAlloc(&h_B, N*sizeof(float), hipHostMallocDefault);
    
    hipMalloc(&d_A, N*sizeof(float));
    hipMalloc(&d_B, N*sizeof(float));
    hipMalloc(&d_result, sizeof(float));
    
    for (int block_size = 32; block_size <= 256; block_size *= 2) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        float total_time = 0;
        for (int i = 0; i < ITERATIONS; ++i) {
            hipMemcpy(d_A, h_A, N*sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_B, h_B, N*sizeof(float), hipMemcpyHostToDevice);
            hipMemset(d_result, 0, sizeof(float));
            
            hipEventRecord(start);
            int grid_size = (N + block_size - 1) / block_size;
            dot_product_kernel<<<grid_size, block_size>>>(d_A, d_B, d_result, N);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            total_time += ms;
        }
        std::cout << "Block size " << block_size 
                  << " average time: " << total_time/ITERATIONS << " ms" << std::endl;
        
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_result);
}

int main() {
    std::cout << "=== Memory Copy Benchmark ===" << std::endl;
    std::cout << "Testing normal memory:" << std::endl;
    benchmark_copy(SIZE * sizeof(float), false);
    
    std::cout << "\nTesting pinned memory:" << std::endl;
    benchmark_copy(SIZE * sizeof(float), true);
    
    std::cout << "\n=== Vector Addition Optimization ===" << std::endl;
    benchmark_vector_add();
    
    std::cout << "\n=== Dot Product Optimization ===" << std::endl;
    benchmark_dot_product();
    
    return 0;
}