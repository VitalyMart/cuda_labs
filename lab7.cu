#include <hip/hip_runtime.h>

#include <iostream>
#include <cmath>

#define PI 3.14159265358979323846
#define THETA_RESOLUTION 360
#define PHI_RESOLUTION 180

// Константная память
__constant__ int c_theta_resolution;
__constant__ int c_phi_resolution;

// Функция на сфере (для GPU)
__device__ float func_on_sphere(float theta, float phi) {
    return sinf(theta) * cosf(phi);
}

// Функция на сфере (для CPU)
float func_on_sphere_cpu(float theta, float phi) {
    return sinf(theta) * cosf(phi);
}

// Заполнение массива значениями функции
void fill_function_data(float *h_data) {
    for (int i = 0; i < THETA_RESOLUTION; ++i) {
        for (int j = 0; j < PHI_RESOLUTION; ++j) {
            float theta = (i * PI) / (THETA_RESOLUTION - 1);
            float phi = (j * 2 * PI) / (PHI_RESOLUTION - 1);
            h_data[i * PHI_RESOLUTION + j] = func_on_sphere_cpu(theta, phi);
        }
    }
}

// Кернел для интеграции с текстурной памятью
__global__ void calculate_integral_texture(hipTextureObject_t tex, float *result) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < c_theta_resolution && y < c_phi_resolution) {
        float theta = (x * PI) / (c_theta_resolution - 1);
        float phi = (y * 2 * PI) / (c_phi_resolution - 1);

        float dtheta = PI / (c_theta_resolution - 1);
        float dphi = (2 * PI) / (c_phi_resolution - 1);

        float value = tex2D<float>(tex, x, y);
        value *= sinf(theta) * dtheta * dphi;

        atomicAdd(result, value);
    }
}

// Кернел для интеграции без текстурной памяти (ступенчатая интерполяция)
__global__ void calculate_integral_no_texture_step(float *result, float *data) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < c_theta_resolution && y < c_phi_resolution) {
        int idx = x * c_phi_resolution + y;
        
        float theta = (x * PI) / (c_theta_resolution - 1);
        float phi = (y * 2 * PI) / (c_phi_resolution - 1);

        float dtheta = PI / (c_theta_resolution - 1);
        float dphi = (2 * PI) / (c_phi_resolution - 1);

        float value = data[idx] * sinf(theta) * dtheta * dphi;

        atomicAdd(result, value);
    }
}

// Кернел для интеграции без текстурной памяти (линейная интерполяция)
__global__ void calculate_integral_no_texture_linear(float *result, float *data) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < c_theta_resolution - 1 && y < c_phi_resolution - 1) {
        int idx1 = x * c_phi_resolution + y;
        int idx2 = (x + 1) * c_phi_resolution + y;
        int idx3 = x * c_phi_resolution + (y + 1);
        int idx4 = (x + 1) * c_phi_resolution + (y + 1);

        float fx1 = data[idx1];
        float fx2 = data[idx2];
        float fy1 = data[idx3];
        float fy2 = data[idx4];

        float theta = (x * PI) / (c_theta_resolution - 1);
        float phi = (y * 2 * PI) / (c_phi_resolution - 1);

        float dtheta = PI / (c_theta_resolution - 1);
        float dphi = (2 * PI) / (c_phi_resolution - 1);

        float alpha = (x + 0.5f) / float(c_theta_resolution - 1);
        float beta = (y + 0.5f) / float(c_phi_resolution - 1);

        float interpolated_value = (1 - alpha) * (1 - beta) * fx1 +
                                   alpha * (1 - beta) * fx2 +
                                   (1 - alpha) * beta * fy1 +
                                   alpha * beta * fy2;

        interpolated_value *= sinf(theta) * dtheta * dphi;

        atomicAdd(result, interpolated_value);
    }
}

int main() {
    float *d_result;
    float h_result = 0.0f;
    hipMalloc((void**)&d_result, sizeof(float));
    hipMemset(d_result, 0, sizeof(float));

    int h_theta_resolution = THETA_RESOLUTION;
    int h_phi_resolution = PHI_RESOLUTION;
    hipMemcpyToSymbol(HIP_SYMBOL(c_theta_resolution), &h_theta_resolution, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_phi_resolution), &h_phi_resolution, sizeof(int));

    float *h_data = new float[THETA_RESOLUTION * PHI_RESOLUTION];
    fill_function_data(h_data);

    float *d_data;
    hipMalloc((void**)&d_data, THETA_RESOLUTION * PHI_RESOLUTION * sizeof(float));
    hipMemcpy(d_data, h_data, THETA_RESOLUTION * PHI_RESOLUTION * sizeof(float), hipMemcpyHostToDevice);

    hipTextureObject_t tex;
    hipArray *d_array;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipMallocArray(&d_array, &channelDesc, THETA_RESOLUTION, PHI_RESOLUTION);
    hipMemcpyToArray(d_array, 0, 0, h_data, THETA_RESOLUTION * PHI_RESOLUTION * sizeof(float), hipMemcpyHostToDevice);

    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = d_array;

    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;

    hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((THETA_RESOLUTION + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (PHI_RESOLUTION + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    calculate_integral_texture<<<numBlocks, threadsPerBlock>>>(tex, d_result);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time with texture memory: " << elapsedTime << " ms" << std::endl;

    hipMemset(d_result, 0, sizeof(float));

    hipEventRecord(start, 0);
    calculate_integral_no_texture_step<<<numBlocks, threadsPerBlock>>>(d_result, d_data);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time without texture memory (step interpolation): " << elapsedTime << " ms" << std::endl;

    hipMemset(d_result, 0, sizeof(float));

    hipEventRecord(start, 0);
    calculate_integral_no_texture_linear<<<numBlocks, threadsPerBlock>>>(d_result, d_data);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time without texture memory (linear interpolation): " << elapsedTime << " ms" << std::endl;

    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Calculated integral: " << h_result << std::endl;

    hipDestroyTextureObject(tex);
    hipFreeArray(d_array);
    hipFree(d_data);
    hipFree(d_result);
    delete[] h_data;

    return 0;
}
